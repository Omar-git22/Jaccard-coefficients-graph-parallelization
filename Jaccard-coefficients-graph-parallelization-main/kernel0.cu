#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"

#define BLOCK_DIM 1024
#define FACTOR 32

__global__ void jaccard_kernel(CSRGraph* csrGraph,  COOMatrix* cooMatrix, unsigned int* numCommonNeighbors, unsigned int* neighborsOfNeighbors){
    __shared__ unsigned int numNeighborsOfNeighbors;
    unsigned int startIndex = blockIdx.x * csrGraph->numVertices;

    if(threadIdx.x==0) {
        numNeighborsOfNeighbors=0;
    }
    __syncthreads();

    for(unsigned int v = 0; v < FACTOR; ++v) {
        unsigned int vertex = FACTOR*blockIdx.x+v;

        if(vertex < csrGraph->numVertices){
            for(unsigned int e = csrGraph->srcPtrs[vertex]; e < csrGraph->srcPtrs[vertex + 1]; e += blockDim.x){
                unsigned int edge = e + threadIdx.x;
                if(edge < csrGraph->srcPtrs[vertex + 1]){
                    unsigned int neighbor = csrGraph->dst[edge];
                    for(unsigned int neighborEdge = csrGraph->srcPtrs[neighbor + 1]; neighborEdge > csrGraph->srcPtrs[neighbor]; --neighborEdge) {
                        unsigned int neighborOfNeighbor = csrGraph->dst[neighborEdge - 1];
                        if(neighborOfNeighbor > vertex) {
                            unsigned int oldVal = atomicAdd(&(numCommonNeighbors[startIndex + neighborOfNeighbor]), 1);
                            if(oldVal == 0) {
                                neighborsOfNeighbors[startIndex + atomicAdd(&(numNeighborsOfNeighbors), 1)] = neighborOfNeighbor;
                            }
                        } else {
                            break;
                        }
                    
                }
            }
        }

        __syncthreads();

        for(unsigned int i = 0; i < numNeighborsOfNeighbors; i += blockDim.x) {
            if(i + threadIdx.x < numNeighborsOfNeighbors){
                unsigned int vertex2 = neighborsOfNeighbors[startIndex + i + threadIdx.x];   
                if(numCommonNeighbors[startIndex + vertex2] > 0) {
                    unsigned int numNeighbors = csrGraph->srcPtrs[vertex + 1] - csrGraph->srcPtrs[vertex];
                    unsigned int numNeighbors2 = csrGraph->srcPtrs[vertex2 + 1] - csrGraph->srcPtrs[vertex2];
                    float jaccardSimilarity = ((float) numCommonNeighbors[startIndex + vertex2])/(numNeighbors + numNeighbors2 - numCommonNeighbors[startIndex + vertex2]);
                    unsigned int j = atomicAdd(&(cooMatrix->nnz), 1);
                    cooMatrix->rowIdxs[j] = vertex;
                    cooMatrix->colIdxs[j] = vertex2;
                    cooMatrix->values[j] = jaccardSimilarity;
                    numCommonNeighbors[startIndex + vertex2] = 0;
                    }
                }
            }
        }
            __syncthreads();
        if(threadIdx.x==0) numNeighborsOfNeighbors=0;
    }
}

void jaccard_gpu0(CSRGraph* csrGraph, CSRGraph* csrGraph_d, COOMatrix* cooMatrix_d) {
    Timer timer;

    // Configurations
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numBlocks = ( csrGraph->numVertices + FACTOR - 1 ) / FACTOR;

    //allocate mem
    startTime(&timer);
    unsigned int* numCommonNeighbors;
    unsigned int* neighborsOfNeighbors;
    hipMalloc((void**) &numCommonNeighbors, numBlocks*csrGraph->numVertices*sizeof(unsigned int)+1);
    hipMalloc((void**) &neighborsOfNeighbors, numBlocks*csrGraph->numVertices*sizeof(unsigned int)+1);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Arrays allocation time");

    //Call Kernel.cu
    startTime(&timer);
    jaccard_kernel <<< numBlocks, numThreadsPerBlock >>> (csrGraph_d, cooMatrix_d, numCommonNeighbors, neighborsOfNeighbors);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

}