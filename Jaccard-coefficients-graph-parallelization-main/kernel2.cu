#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"
#include <limits.h>

#define BLOCK_DIM 1024
#define FACTOR 32
#define BATCH 32
#define ITER 10
#define SMEM_SIZE ITER*BLOCK_DIM
#define ANSI_COLOR_MAGENTA    "\x1b[35m"

__global__ void jaccard_kernel2(CSRGraph* csrGraph,  COOMatrix* cooMatrix, unsigned int* numCommonNeighbors, unsigned int* neighborsOfNeighbors){
    __shared__ unsigned int numCommonNeighbors_s[SMEM_SIZE];
    __shared__ unsigned int numNeighborsOfNeighbors;
    
    unsigned int startIndex = blockIdx.x * csrGraph->numVertices;

    if(threadIdx.x==0) {
        numNeighborsOfNeighbors=0;
    }
    for(int i=0; i < ITER; ++i){
        numCommonNeighbors_s[i * BLOCK_DIM + threadIdx.x] = 0;
    }
    __syncthreads();
    for(unsigned int v = 0; v < FACTOR; ++v) {
        unsigned int vertex = FACTOR * blockIdx.x + v;
        if(vertex < csrGraph->numVertices) {
            for(unsigned int e = csrGraph->srcPtrs[vertex]; e < csrGraph->srcPtrs[vertex + 1]; e += blockDim.x / BATCH){
                unsigned int edge = e + threadIdx.x / BATCH;
                if(edge < csrGraph->srcPtrs[vertex + 1]) {
                    int neighbor = csrGraph->dst[edge];
                    for(long neighborEdge = (long) csrGraph->srcPtrs[neighbor + 1]; neighborEdge > (long) csrGraph->srcPtrs[neighbor]; neighborEdge -= BATCH ) {
                        if( neighborEdge - ( threadIdx.x % BATCH ) - 1 > csrGraph->srcPtrs[neighbor]) {
                            unsigned int neighborOfNeighbor = csrGraph->dst[neighborEdge - ( threadIdx.x % BATCH ) - 1];
                            if(neighborOfNeighbor > vertex) {
                                unsigned int oldVal;
                                if(neighborOfNeighbor < SMEM_SIZE) {
                                    oldVal = atomicAdd(&(numCommonNeighbors_s[neighborOfNeighbor]), 1);
                                } else {
                                    oldVal = atomicAdd(&(numCommonNeighbors[startIndex + neighborOfNeighbor]), 1);
                                }
                                if( oldVal == 0 ) {
                                    neighborsOfNeighbors[startIndex + atomicAdd(&(numNeighborsOfNeighbors), 1)] = neighborOfNeighbor;
                                }
                            } else {
                                break;
                            }
                        }
                    }
                }
            }
            __syncthreads();
            for(unsigned int i = 0; i < numNeighborsOfNeighbors; i += blockDim.x) {
                if(i + threadIdx.x < numNeighborsOfNeighbors){
                    unsigned int vertex2;
                    vertex2 = neighborsOfNeighbors[startIndex + i + threadIdx.x]; 
                    if(vertex2 < SMEM_SIZE){
                        if(numCommonNeighbors_s[vertex2] > 0) {
                            unsigned int numNeighbors = csrGraph->srcPtrs[vertex + 1] - csrGraph->srcPtrs[vertex];
                            unsigned int numNeighbors2 = csrGraph->srcPtrs[vertex2 + 1] - csrGraph->srcPtrs[vertex2];
                            float jaccardSimilarity = ((float) numCommonNeighbors_s[vertex2])/(numNeighbors + numNeighbors2 - numCommonNeighbors_s[vertex2]);
                            unsigned int j = atomicAdd(&(cooMatrix->nnz), 1);
                            cooMatrix->rowIdxs[j] = vertex;
                            cooMatrix->colIdxs[j] = vertex2;
                            cooMatrix->values[j] = jaccardSimilarity;
                            numCommonNeighbors_s[vertex2] = 0;
                        }
                    }else{
                        if(numCommonNeighbors[startIndex + vertex2] > 0) {
                            unsigned int numNeighbors = csrGraph->srcPtrs[vertex + 1] - csrGraph->srcPtrs[vertex];
                            unsigned int numNeighbors2 = csrGraph->srcPtrs[vertex2 + 1] - csrGraph->srcPtrs[vertex2];
                            float jaccardSimilarity = ((float) numCommonNeighbors[startIndex + vertex2])/(numNeighbors + numNeighbors2 - numCommonNeighbors[startIndex + vertex2]);
                            unsigned int j = atomicAdd(&(cooMatrix->nnz), 1);
                            cooMatrix->rowIdxs[j] = vertex;
                            cooMatrix->colIdxs[j] = vertex2;
                            cooMatrix->values[j] = jaccardSimilarity;
                            numCommonNeighbors[startIndex + vertex2] = 0;
                        }
                    }         
                }
            }
        }
        __syncthreads();
        if(threadIdx.x==0) numNeighborsOfNeighbors=0;
    }
}

void jaccard_gpu2(CSRGraph* csrGraph, CSRGraph* csrGraph_d, COOMatrix* cooMatrix_d) {
    Timer timer;

    // Configurations
    const unsigned int numThreadsPerBlock = BLOCK_DIM;
    const unsigned int numBlocks = ( csrGraph->numVertices + FACTOR - 1 ) / FACTOR;

    //allocate mem
    startTime(&timer);
    unsigned int* numCommonNeighbors;
    unsigned int* neighborsOfNeighbors;
    hipMalloc((void**) &numCommonNeighbors, numBlocks*csrGraph->numVertices*sizeof(unsigned int)+1);
    hipMalloc((void**) &neighborsOfNeighbors, numBlocks*csrGraph->numVertices*sizeof(unsigned int)+1);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Arrays allocation time");

    //Call Kernel.cu
    startTime(&timer);
    jaccard_kernel2 <<< numBlocks, numThreadsPerBlock >>> (csrGraph_d, cooMatrix_d, numCommonNeighbors, neighborsOfNeighbors);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);
}